#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 20
#define STREAMS 4

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    hipStream_t streams[STREAMS];

    // Allocate host memory
    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    h_c = (float*)malloc(N * sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i+1;
        h_b[i] = i+1;
    }
    printf("A array:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f\n", h_a[i]);
    }
    printf("B array:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f\n", h_b[i]);
    }

    // Create streams
    for (int i = 0; i < STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Divide work among streams
    int shared_mem = 0;
    int threads_per_block = 256;
    int streamSize = N / STREAMS;
    int n_blocks = (streamSize + threads_per_block - 1) / threads_per_block;
    for (int i = 0; i < STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_b[offset], &h_b[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        
        vector_add<<<(n_blocks, threads_per_block, shared_mem, streams[i]>>>(&d_a[offset], &d_b[offset], &d_c[offset], streamSize);
        
        hipMemcpyAsync(&h_c[offset], &d_c[offset], streamSize * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    // Wait for all streams to complete
    hipDeviceSynchronize();

    printf("C array:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f\n", h_c[i]);
    }

    // Clean up
    for (int i = 0; i < STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
