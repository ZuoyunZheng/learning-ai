#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <cstdio>

#define MATRIX_SIZE 16
#define WMMA_TILE_SIZE 16

using namespace nvcuda::wmma;

// This is the CUDA kernel
__global__ void wmma_example_kernel(const half* a, const half* b, half* c) {
    // Define the fragment types for the input and output matrices
    fragment<matrix_a, WMMA_TILE_SIZE, WMMA_TILE_SIZE, WMMA_TILE_SIZE, half, row_major> a_frag;
    fragment<matrix_b, WMMA_TILE_SIZE, WMMA_TILE_SIZE, WMMA_TILE_SIZE, half, col_major> b_frag;
    fragment<accumulator, WMMA_TILE_SIZE, WMMA_TILE_SIZE, WMMA_TILE_SIZE, half> c_frag;
    // Initialize the output to zero
    fill_fragment(c_frag, __float2half(0.0f));

    // Load the input matrices into the fragments
    load_matrix_sync(a_frag, a, MATRIX_SIZE);
    load_matrix_sync(b_frag, b, MATRIX_SIZE);

    // Perform the matrix multiplication
    mma_sync(c_frag, a_frag, b_frag, c_frag);

    // Store the result back to memory
    store_matrix_sync(c, c_frag, MATRIX_SIZE, mem_row_major);
}

int main() {
    printf("Warp-level Matrix Muliply Accumulate example.\n\n");
    // Initialize host matrices
    half h_a[MATRIX_SIZE * MATRIX_SIZE];
    half h_b[MATRIX_SIZE * MATRIX_SIZE];
    half h_c[MATRIX_SIZE * MATRIX_SIZE];

    // Fill matrices with example data
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i) {
        h_a[i] = __float2half(1.0f);
        h_b[i] = __float2half(1.0f);
    }

    // Allocate device memory
    half *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, MATRIX_SIZE * MATRIX_SIZE * sizeof(half));
    hipMalloc((void**)&d_b, MATRIX_SIZE * MATRIX_SIZE * sizeof(half));
    hipMalloc((void**)&d_c, MATRIX_SIZE * MATRIX_SIZE * sizeof(half));

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, MATRIX_SIZE * MATRIX_SIZE * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, MATRIX_SIZE * MATRIX_SIZE * sizeof(half), hipMemcpyHostToDevice);

    // Launch the WMMA kernel
    wmma_example_kernel<<<1, 32>>>(d_a, d_b, d_c);

    // Copy the result back to host
    hipMemcpy(h_c, d_c, MATRIX_SIZE * MATRIX_SIZE * sizeof(half), hipMemcpyDeviceToHost);

    // Print the result matrix
    for (int i = 0; i < MATRIX_SIZE; ++i) {
        for (int j = 0; j < MATRIX_SIZE; ++j) {
            printf("%.0f ", __half2float(h_c[i * MATRIX_SIZE + j]));
        }
	printf("\n");
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
