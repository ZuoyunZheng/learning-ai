#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 20
#define STREAMS 4

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    // Allocate host memory
    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    h_c = (float*)malloc(N * sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i+1;
        h_b[i] = i+1;
    }

    // Create stream
    hipStreamCreate(&stream);

    // Begin graph capture
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Add operations to the graph
    int streamSize = N / STREAMS;
    int shared_mem = 0;
    int block_dim = 256;
    int grid_dim = (streamSize + 255) / block_dim;

    for (int i = 0; i < STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(&d_b[offset], &h_b[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, stream);

        vector_add<<<grid_dim, block_dim, shared_mem, stream>>>(&d_a[offset], &d_b[offset], &d_c[offset], streamSize);

        hipMemcpyAsync(&h_c[offset], &d_c[offset], streamSize * sizeof(float), hipMemcpyDeviceToHost, stream);
    }

    // End graph capture
    hipStreamEndCapture(stream, &graph);

    // Create executable graph
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    // Launch the graph
    hipGraphLaunch(instance, stream);

    // Wait for the graph to complete
    hipStreamSynchronize(stream);

    // Print results
    printf("Result array C:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f\n", h_c[i]);
    }

    // Clean up
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
