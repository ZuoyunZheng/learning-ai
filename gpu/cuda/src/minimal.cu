#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void myKernel() {}

int main() {
    hipError_t err = hipSuccess;
    int count = 0;

    int runtimeVersion = 0;

    err = hipRuntimeGetVersion(&runtimeVersion);
    if (err != hipSuccess) {
        printf("hipRuntimeGetVersion failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("CUDA Runtime version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    int driverVersion = 0;
    err = hipDriverGetVersion(&driverVersion);
    if (err != hipSuccess) {
        printf("hipDriverGetVersion failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("CUDA Driver version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA device count: %d\n", count);

    // Get and print total VRAM for each device
    for (int i = 0; i < count; i++) {
        hipSetDevice(i);
        size_t free_mem, total_mem;
        err = hipMemGetInfo(&free_mem, &total_mem);
        if (err != hipSuccess) {
            printf("hipMemGetInfo failed for device %d: %s\n", i, hipGetErrorString(err));
        } else {
            printf("Device %d - Total VRAM: %.2f GB\n", i, total_mem / (1024.0 * 1024.0 * 1024.0));
        }
    }

    myKernel<<<1,1>>>();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA program ran successfully\n");
    return 0;
}
