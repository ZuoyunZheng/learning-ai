
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() {}

int main() {
    hipError_t err = hipSuccess;
    int count = 0;

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA device count: %d\n", count);

    myKernel<<<1,1>>>();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA program ran successfully\n");
    return 0;
}
