#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() {}

int main() {
    hipError_t err = hipSuccess;
    int count = 0;

    int runtimeVersion = 0;

    err = hipRuntimeGetVersion(&runtimeVersion);
    if (err != hipSuccess) {
        printf("hipRuntimeGetVersion failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("CUDA Runtime version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    int driverVersion = 0;
    err = hipDriverGetVersion(&driverVersion);
    if (err != hipSuccess) {
        printf("hipDriverGetVersion failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("CUDA Driver version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA device count: %d\n", count);

    myKernel<<<1,1>>>();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA program ran successfully\n");
    return 0;
}
