
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() {}

int main() {
    hipError_t err = hipSuccess;
    int count = 0;

    int runtimeVersion = 0;

    // Get CUDA runtime version
    err = hipRuntimeGetVersion(&runtimeVersion);
    if (err != hipSuccess) {
        printf("hipRuntimeGetVersion failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA Runtime version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);


    err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA device count: %d\n", count);

    myKernel<<<1,1>>>();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA program ran successfully\n");
    return 0;
}
